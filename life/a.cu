#include "hip/hip_runtime.h"
#include <iostream>
#include <random>
#include <SDL.h>
using namespace std;

using num_t = uint16_t;
using num_udist = uniform_int_distribution<num_t>;

num_t scr_w = 2560;
num_t scr_h = 1440;
num_t life_w = 32;
num_t life_h = 32;
num_t food_w = 4;
num_t food_h = 4;

#define life_num 128
#define food_num 128

struct life_t {
    num_t x;
    num_t y;
    float energy;
} lifes[life_num];

struct food_t {
    num_t x;
    num_t y;
} foods[food_num];

mt19937 gen(1);

void init_lifes() {
    num_udist x_dist(0, scr_w - life_w);
    num_udist y_dist(0, scr_h - life_h);
    for (num_t i = 0; i < life_num; i++) {
        lifes[i].x = x_dist(gen);
        lifes[i].y = y_dist(gen);
        lifes[i].energy = 0;
    }
}

num_udist food_x_dist(0, scr_w - food_w);
num_udist food_y_dist(0, scr_h - food_h);

void init_foods() {
    for (num_t i = 0; i < food_num; i++) {
        foods[i].x = food_x_dist(gen);
        foods[i].y = food_y_dist(gen);
    }
}

SDL_Event event;
SDL_Window* window;
SDL_Renderer* rendr;

int init_gfx() {
    if (SDL_Init(SDL_INIT_VIDEO) < 0) {
        fprintf(stderr, "Couldn't intialize SDL: %s\n", SDL_GetError());
        return EXIT_FAILURE;
    }
    window = SDL_CreateWindow("block-life", SDL_WINDOWPOS_UNDEFINED,
        SDL_WINDOWPOS_UNDEFINED, scr_w, scr_h,
        SDL_WINDOW_BORDERLESS | SDL_WINDOW_SHOWN);
    if (!window) {
        fprintf(stderr, "Couldn't create SDL window: %s\n", SDL_GetError());
        SDL_Quit();
        return EXIT_FAILURE;
    }
    rendr = SDL_CreateRenderer(window, -1, 0);
    return EXIT_SUCCESS;
}

void draw_sq(num_t x1, num_t y1, num_t w, num_t h) {
    for (num_t x = x1; x < x1 + w; x++) {
        SDL_RenderDrawPoint(rendr, x, y1);
        SDL_RenderDrawPoint(rendr, x, y1 + h - 1);
    }
    for (num_t y = y1 + 1; y < y1 + h - 1; y++) {
        SDL_RenderDrawPoint(rendr, x1, y);
        SDL_RenderDrawPoint(rendr, x1 + w - 1, y);
    }
}

void show() {
    SDL_SetRenderDrawColor(rendr, 255, 255, 255, 255);
    SDL_RenderClear(rendr);
    SDL_SetRenderDrawColor(rendr,   0,   0,   0, 255);
    for (num_t i = 0; i < food_num; i++) {
        draw_sq(foods[i].x, foods[i].y, food_w, food_h);
    }
    for (num_t i = 0; i < life_num; i++) {
        draw_sq(lifes[i].x, lifes[i].y, life_w, life_h);
    }
    SDL_RenderPresent(rendr);
}

void move() {
    num_udist dist(0, 2);
    for (num_t i = 0; i < life_num; i++) {
        lifes[i].x += dist(gen);
        if (lifes[i].x > 0) {
            if (lifes[i].x < scr_w) {
                lifes[i].x--;
            } else {
                lifes[i].x = scr_w - 1;
            }
        }
        lifes[i].y += dist(gen);
        if (lifes[i].y > 0) {
            if (lifes[i].y < scr_h) {
                lifes[i].y--;
            } else {
                lifes[i].y = scr_h - 1;
            }
        }
        for (num_t j = 0; j < food_num; j++) {
            if (foods[j].x > lifes[i].x &&
                foods[j].x + food_w < lifes[i].x + life_w &&
                foods[j].y > lifes[i].y &&
                foods[j].y + food_h < lifes[i].y + life_h) {

                lifes[i].energy++;
                foods[j].x = food_x_dist(gen);
                foods[j].y = food_y_dist(gen);
            }
        }
    }
}

int main() {
    init_lifes();
    init_foods();
    if (int r = init_gfx()) {
        return r;
    }
    while (true) {
        show();
        while (SDL_PollEvent(&event)) {
            if (event.type == SDL_QUIT) {
                SDL_DestroyRenderer(rendr);
                SDL_DestroyWindow(window);
                SDL_Quit();
                printf("Exited normally.\n");
                return EXIT_SUCCESS;
            }
        }
        move();
    }
}
